#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <random>
#include <vector>
#include <cassert>
#include "gemm_comm.cu"

// Naive GEMM
__global__ void gemm_naive(float *dA, float *dB, float *dC, int M, int K, int N)
{
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < M && col < N) {
        float tmp = 0;
        for (int k=0;k<K;++k) {
            tmp += dA[row * K + k] * dB[k * N + col];
        }
        dC[row * N + col] = tmp;
    }
}

// shared mem with tile
template<int TILE_SIZE> __global__ void gemm_shared(float *dA, float *dB, float *dC, int M, int K, int N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int width = (K + TILE_SIZE - 1) / TILE_SIZE;

    __shared__ float SA[TILE_SIZE][TILE_SIZE];
    __shared__ float SB[TILE_SIZE][TILE_SIZE];
    float reg_tmp = 0;

    for (int w=0;w<width;++w) {
        if (row < M && (w * TILE_SIZE + threadIdx.y) < K) {
            SA[threadIdx.x][threadIdx.y] = dA[row * K + w * TILE_SIZE + threadIdx.y];
        } else {
            SA[threadIdx.x][threadIdx.y] = 0;
        }
        if (col < N && (w * TILE_SIZE + threadIdx.x) < K) {
            SB[threadIdx.x][threadIdx.y] = dB[(w * TILE_SIZE + threadIdx.x)*N + col];
        } else {
            SB[threadIdx.x][threadIdx.y] = 0;
        }
        __syncthreads();

        for (int s=0;s<TILE_SIZE;++s) {
            reg_tmp += SA[threadIdx.x][s] * SB[s][threadIdx.y];
        }
        __syncthreads();
    }
    if (row < M && col < N) {
        dC[row * N + col] = reg_tmp;
    }
    
}

template<int TILE_SIZE>
__global__ void gemm_shared_transposed(float *dA, float *dB, float *dC, int M, int K, int N) {
    // Block indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Starting indices for this block
    int row = TILE_SIZE * bx + tx;
    int col = TILE_SIZE * by + ty;
    
    // Shared memory tiles
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load A tile - directly in row-major order
        if (row < M && (t * TILE_SIZE + ty) < K) {
            As[tx][ty] = dA[row * K + t * TILE_SIZE + ty];
        } else {
            As[tx][ty] = 0.0f;
        }
        
        // Load B tile - with transposition
        if ((t * TILE_SIZE + tx) < K && col < N) {
            // Original B is in row-major: B[k][n]
            // Load it transposed into shared memory
            Bs[ty][tx] = dB[(t * TILE_SIZE + tx) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute on the tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            // Now both matrices are accessed in a coalesced manner
            sum += As[tx][k] * Bs[ty][k];
        }
        
        __syncthreads();
    }
    
    // Store result
    if (row < M && col < N) {
        dC[row * N + col] = sum;
    }
}

int main() {
    
    int m = 1024;
    int n = 1024;
    int k = 1024;

    int trials = 100;

    int matrixSize = m * n;

    thrust::host_vector<float> h_A(matrixSize);
    thrust::host_vector<float> h_B(matrixSize);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0, 10);

    for (int i = 0; i < matrixSize; ++i) {
        h_A[i] = dis(gen);
        h_B[i] = dis(gen);
    }

    thrust::device_vector<float> d_A = h_A;
    thrust::device_vector<float> d_B = h_B;
    thrust::device_vector<float> d_C(matrixSize, 0.0);

    dim3 threadNum(32, 32);
    dim3 blockNum((m + threadNum.x - 1)/threadNum.x, (n + threadNum.y - 1)/threadNum.y);

    float ker_time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i=0;i<trials;++i) {
        //gemm_naive<<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, k, n);
        //gemm_shared<32><<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, k, n);
        gemm_shared_transposed<32><<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, k, n);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);
    printf("kernel time: %.4f second, %.4f ms\n", ker_time / (trials * 1000.), ker_time / trials);
    printf("grid dim: %d, %d, %d\n", blockNum.x, blockNum.y, blockNum.z);
    printf("block dim: %d, %d, %d\n", threadNum.x, threadNum.y, threadNum.z);

    thrust::host_vector<float> h_C = d_C;

    std::vector<float> cpu_v(m*n);
    double st, ela;
    st = get_walltime();
    matrixSerial(h_A.data(), h_B.data(), cpu_v.data(), m, k, n);
    ela = get_walltime() - st;
    printf("CPU time:%.2f second\n", ela);

    compare(h_C.data(), cpu_v.data(), m, n);
    
    return 0;
}