#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <random>
#include <vector>
#include <cassert>
#include "gemm_comm.cu"

// Naive GEMM
__global__ void gemm_naive(float *dA, float *dB, float *dC, int M, int K, int N)
{
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < M && col < N) {
        float tmp = 0;
        for (int k=0;k<K;++k) {
            tmp += dA[row * K + k] * dB[k * N + col];
        }
        dC[row * N + col] = tmp;
    }
}

// shared mem with tile
template<int TILE_SIZE> __global__ void gemm_shared(float *dA, float *dB, float *dC, int M, int K, int N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int width = (K + TILE_SIZE - 1) / TILE_SIZE;

    __shared__ float SA[TILE_SIZE][TILE_SIZE];
    __shared__ float SB[TILE_SIZE][TILE_SIZE];
    float reg_tmp = 0;

    for (int w=0;w<width;++w) {
        if (row < M && (w * TILE_SIZE + threadIdx.y) < K) {
            SA[threadIdx.x][threadIdx.y] = dA[row * K + w * TILE_SIZE + threadIdx.y];
        } else {
            SA[threadIdx.x][threadIdx.y] = 0;
        }
        if (col < N && (w * TILE_SIZE + threadIdx.x) < K) {
            SB[threadIdx.x][threadIdx.y] = dB[(w * TILE_SIZE + threadIdx.x)*N + col];
        } else {
            SB[threadIdx.x][threadIdx.y] = 0;
        }
        __syncthreads();

        for (int s=0;s<TILE_SIZE;++s) {
            reg_tmp += SA[threadIdx.x][s] * SB[s][threadIdx.y];
        }
        __syncthreads();
    }
    if (row < M && col < N) {
        dC[row * N + col] = reg_tmp;
    }
}

template<int TILE_SIZE> __global__ void gemm_reg(const float *dA, const float *dB, float *dC, int M, int K, int N) {

    int c = threadIdx.x;
    int r = threadIdx.y;
    
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    __shared__ float SA[TILE_SIZE][TILE_SIZE];
    __shared__ float SB[TILE_SIZE][TILE_SIZE];

    float reg_tile = 0;
    for (int t = 0; t < K; t += TILE_SIZE) {

        if (row < M && (t + c) < K) {
            SA[r][c] = dA[row * K + (t + c)];
        } else {
            SA[r][c] = 0;
        }

        if ((t + r) < K && col < N) {
            SB[r][c] = dB[(t + r) * N + col];
        } else {
            SB[r][c] = 0;
        }
        __syncthreads();

        // accumulate sum
        // global idx = i * N + j;
        for (int k = 0; k < TILE_SIZE; ++k) {
            reg_tile += SA[r][k] * SB[k][c];
        }
        __syncthreads();
    }

    if (row < M && col < N) {
        dC[row * N + col] = reg_tile;
    }
    
}

template<int TILE_SIZE>
__global__ void gemm_shared_transposed(float *dA, float *dB, float *dC, int M, int K, int N) {
    // Block indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Starting indices for this block
    int row = TILE_SIZE * bx + tx;
    int col = TILE_SIZE * by + ty;
    
    // Shared memory tiles
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load A tile - directly in row-major order
        if (row < M && (t * TILE_SIZE + ty) < K) {
            As[tx][ty] = dA[row * K + t * TILE_SIZE + ty];
        } else {
            As[tx][ty] = 0.0f;
        }
        
        // Load B tile - with transposition
        if ((t * TILE_SIZE + tx) < K && col < N) {
            // Original B is in row-major: B[k][n]
            // Load it transposed into shared memory
            Bs[ty][tx] = dB[(t * TILE_SIZE + tx) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute on the tile
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            // Now both matrices are accessed in a coalesced manner
            sum += As[tx][k] * Bs[ty][k];
        }
        
        __syncthreads();
    }
    
    // Store result
    if (row < M && col < N) {
        dC[row * N + col] = sum;
    }
}

template<int TILE_SIZE>
__global__ void gemm_three(const float *A, const float *B, const float *C, float *D, int M, int N, int K, int L) {
    // M*K, K*N, N*L
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int r = threadIdx.y;
    int c = threadIdx.x;
    
    __shared__ float SA[TILE_SIZE][TILE_SIZE];
    __shared__ float SB[TILE_SIZE][TILE_SIZE];
    __shared__ float SC[TILE_SIZE][TILE_SIZE];
    __shared__ float AB[TILE_SIZE][TILE_SIZE];

    float reg = 0;
    for (int n = 0; n < N; n += TILE_SIZE) {
        AB[r][c] = 0;

        for (int k = 0; k < K; k += TILE_SIZE) {
            if (row < M && (k + c) < K) {
                SA[r][c] = A[row * K + (k + c)];
            } else {
                SA[r][c] = 0;
            }
    
            if ((k + r) < K && (n + c) < N) {
                SB[r][c] = B[(k + r) * N + (n + c)];
            } else {
                SB[r][c] = 0.0f;
            }
            __syncthreads();
    
            for (int i = 0; i < TILE_SIZE; ++i) {
                AB[r][c] += SA[r][i] * SB[i][c];
            }
            __syncthreads();
        }

        if ((n + r) < N && col < L) {
            SC[r][c] = C[(n + r) * L + col];
        } else {
            SC[r][c] = 0;
        }
        __syncthreads();

        // 计算(A*B)*C的部分结果
        for (int i = 0; i < TILE_SIZE; ++i) {
            if ((n + i) < N) {
                reg += AB[r][i] * SC[i][c];
            }
        }
        __syncthreads();
    }

    if (row < M && col < L) {
        D[row * L + col] = reg;
    }
    
}

int main() {
    
    int m = 1024;
    int n = 1024;
    int k = 1024;
    int l = 1024;

    int trials = 100;

    int matrixSize = m * n;

    thrust::host_vector<float> h_A(matrixSize);
    thrust::host_vector<float> h_B(matrixSize);
    thrust::host_vector<float> h_C(matrixSize);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0, 10);

    for (int i = 0; i < matrixSize; ++i) {
        h_A[i] = dis(gen);
        h_B[i] = dis(gen);
        h_C[i] = dis(gen);
    }

    thrust::device_vector<float> d_A = h_A;
    thrust::device_vector<float> d_B = h_B;
    thrust::device_vector<float> d_C = h_C;
    thrust::device_vector<float> d_D1(matrixSize, 0.0);

    dim3 threadNum(8, 8);
    dim3 blockNum((m + threadNum.x - 1)/threadNum.x, (n + threadNum.y - 1)/threadNum.y);

    float ker_time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    thrust::device_vector<float> d_tmp(matrixSize, 0.0);
    for (int i=0;i<trials;++i) {
        //gemm_naive<<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, k, n);
        //gemm_shared<32><<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, k, n);
        //gemm_shared_transposed<32><<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, k, n);
        gemm_reg<8><<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_tmp.data().get(), m, k, n);
        gemm_reg<8><<<blockNum, threadNum>>>(d_tmp.data().get(), d_C.data().get(), d_D1.data().get(), m, k, n);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);
    printf("kernel time: %.4f second, %.4f ms\n", ker_time / (trials * 1000.), ker_time / trials);
    printf("grid dim: %d, %d, %d\n", blockNum.x, blockNum.y, blockNum.z);
    printf("block dim: %d, %d, %d\n", threadNum.x, threadNum.y, threadNum.z);

    //---------------CPU

    std::vector<float> tmp_cpu_v(m*n);
    std::vector<float> cpu_v(m*n);
    double st, ela;
    st = get_walltime();
    matrixSerial(h_A.data(), h_B.data(), tmp_cpu_v.data(), m, k, n);
    matrixSerial(tmp_cpu_v.data(), h_C.data(), cpu_v.data(), m, k, n);
    ela = get_walltime() - st;
    printf("CPU time:%.2f second\n", ela);
    //---------------

    thrust::device_vector<float> d_D2(matrixSize, 0.0);
    gemm_three<8><<<blockNum, threadNum>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), d_D2.data().get(), m, n, k, l);

    thrust::host_vector<float> h_D1 = d_D1;
    thrust::host_vector<float> h_D2 = d_D2;
    compare(h_D2.data(), h_D1.data(), m, n);
    
    return 0;
}