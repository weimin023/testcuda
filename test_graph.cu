#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hip/hip_complex.h>

#define CHECK_CUDA(call)                                                                                                                                                                               \
    do {                                                                                                                                                                                               \
        hipError_t err = call;                                                                                                                                                                        \
        if (err != hipSuccess) {                                                                                                                                                                      \
            fprintf(stderr, "Cuda failed with error code %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);                                                                    \
            exit(EXIT_FAILURE);                                                                                                                                                                        \
        }                                                                                                                                                                                              \
    } while (0)

#define CHECK_CUSOLVER(call)                                                                                                                                                                           \
    do {                                                                                                                                                                                               \
        hipsolverStatus_t stat = call;                                                                                                                                                                  \
        if (stat != HIPSOLVER_STATUS_SUCCESS) {                                                                                                                                                         \
            fprintf(stderr, "CuSolver failed with error code %s at line %d in file %s\n", cusolverGetErrorString(stat), __LINE__, __FILE__);                                                           \
            exit(EXIT_FAILURE);                                                                                                                                                                        \
        }                                                                                                                                                                                              \
    } while (0)

const char *cusolverGetErrorString(hipsolverStatus_t error) {
    switch (error) {
    case HIPSOLVER_STATUS_SUCCESS:
        return "CUSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "CUSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "CUSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "CUSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "CUSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "CUSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "CUSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "CUSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "CUSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "CUSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "CUSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    default:
        return "UNKNOWN_ERROR";
    }
}
// Custom CUDA kernel (simple operation)
__global__ void myKernel(hipComplex *data, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        data[idx].x += 1.0f;
    }
}

int main() {
    const int m = 3;
    hipStream_t stream;
    hipsolverHandle_t cusolverH;
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
    CHECK_CUSOLVER(hipsolverSetStream(cusolverH, stream));

    

    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipFloatComplex *d_work;
    int *devInfo;
    int lwork = 0;
    hipFloatComplex *m_eigen_vec, *m_eigen_vec_row_majored;
    float *m_eigen_value;

    CHECK_CUDA(hipMallocAsync(&m_eigen_vec, m*m*sizeof(hipFloatComplex), stream));
    CHECK_CUDA(hipMallocAsync(&m_eigen_vec_row_majored, m*m*sizeof(hipFloatComplex), stream));
    CHECK_CUDA(hipMallocAsync(&m_eigen_value, m*sizeof(float), stream));
    CHECK_CUDA(hipMallocAsync(&devInfo, sizeof(int), stream));

    CHECK_CUSOLVER(hipsolverDnCheevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, m, m_eigen_vec, m, m_eigen_value, &lwork));

    
    CHECK_CUDA(hipMallocAsync(&d_work, lwork * sizeof(hipFloatComplex), stream));
    

    hipFloatComplex h_A[m * m] = {
        {1, 0}, {2, 0}, {3, 0},
        {2, 0}, {5, 0}, {6, 0},
        {3, 0}, {6, 0}, {9, 0}
    };
    hipFloatComplex *d_A;
    CHECK_CUDA(hipMallocAsync(&d_A, m * m * sizeof(hipFloatComplex), stream));
    CHECK_CUDA(hipMemcpyAsync(d_A, h_A, m * m * sizeof(hipFloatComplex), hipMemcpyHostToDevice, stream));
    

    // NODE A: EVD
    CHECK_CUDA(hipMemcpyAsync(m_eigen_vec, d_A, m * m * sizeof(hipFloatComplex), hipMemcpyDeviceToDevice, stream));

    CHECK_CUDA(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
    CHECK_CUSOLVER(hipsolverDnCheevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, m, d_A, m, m_eigen_value, d_work, lwork, devInfo));

    // NODE B: My Kernel
    
    int blockSize = 32;
    int gridSize = (m * m + blockSize - 1) / blockSize;
    myKernel<<<gridSize, blockSize, 0, stream>>>(d_A, m * m);

    CHECK_CUDA(hipStreamEndCapture(stream, &graph));
    CHECK_CUDA(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    // Clean up
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);

    return 0;
}
