#include "hip/hip_runtime.h"
#include "evd.cuh"
#include "../sharedmem.h"

namespace TronBLAS {
    template <typename T> 
    __global__ void transpose_kernal(T *in, T *out, int m, int n) {

        SharedMemory<T> smem;
        T *shm = smem.getPointer();
        
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < m && y < n) {
            shm[x + y*m] = in[x + y*m];
            
            __syncthreads();

            out[x*n + y] = shm[x + y*m];
        }
    }
}

namespace TronCUDA {
    template <> void cuSyEVD<float, float>::preprocess() {
        (hipsolverDnSsyevd_bufferSize(
            cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N,
            m_eigen_vec.data().get(), N, m_eigen_value.data().get(), &lwork));
    
        (hipMalloc(&d_work, lwork * sizeof(float)));
    }
    
    template <> void cuSyEVD<hipComplex, float>::preprocess() {
        (hipsolverDnCheevd_bufferSize(
            cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N,
            m_eigen_vec.data().get(), N, m_eigen_value.data().get(), &lwork));
    
        hipMalloc(&d_work, lwork * sizeof(hipComplex));
    }
    
    template <> void cuSyEVD<hipDoubleComplex, double>::preprocess() {
        (hipsolverDnZheevd_bufferSize(
            cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N,
            m_eigen_vec.data().get(), N, m_eigen_value.data().get(), &lwork));
    
        (hipMalloc(&d_work, lwork * sizeof(hipDoubleComplex)));
    }
    
    template <> void cuSyEVD<hipFloatComplex, float>::exec(const thrust::device_vector<hipFloatComplex> &d_data) {
        m_eigen_vec = d_data;
        (hipsolverDnCheevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N, m_eigen_vec.data().get(), N, m_eigen_value.data().get(), d_work, lwork, devInfo));
    
        int info;
        (hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
        if (info != 0) { std::cerr << "Error: cuSOLVER operation failed with info = " << info << std::endl; }
    
        toRow();
    }
    
    template <> void cuSyEVD<hipDoubleComplex, double>::exec(const thrust::device_vector<hipDoubleComplex> &d_data) {
        m_eigen_vec = d_data;
        (hipsolverDnZheevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N, m_eigen_vec.data().get(), N, m_eigen_value.data().get(), d_work, lwork, devInfo));
    
        int info;
        (hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
        if (info != 0) { std::cerr << "Error: cuSOLVER operation failed with info = " << info << std::endl; }
    
        toRow();
    }
    
    template <typename T, typename D>
    void cuSyEVD<T, D>::toRow() {
        dim3 blockSize(16, 16);
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    
        size_t shm_size = blockSize.x * blockSize.y * sizeof(T);
        TronBLAS::transpose_kernal<<<gridSize, blockSize, shm_size, m_stream>>>(m_eigen_vec.data().get(), m_eigen_vec_row_majored.data().get(), N, N);
    }
}
