#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <vector>
#include <fstream>

// Constants
#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 4
#define WARP_SIZE 32

// Distance operators
__device__ unsigned start_distance(unsigned pixels, unsigned tx) {
    return __clz(~(pixels << (32-tx)));
}

__device__ unsigned end_distance(unsigned pixels, unsigned tx) {
    return __ffs(~(pixels >> (tx+1)));
}

// Build bitmask of pixels in a warp
__device__ unsigned buildBitmask(int value, int tid) {
    // Use ballot_sync to create a bitmask where each bit represents a thread's predicate
    return __ballot_sync(0xFFFFFFFF, value);
}

// Find representative label using path compression
__device__ unsigned find(unsigned* labels, unsigned x) {
    unsigned y = x;
    while (y != labels[y]) {
        y = labels[y];
    }
    
    // Path compression - update all nodes in the path to point to the root
    while (x != y) {
        unsigned tmp = labels[x];
        labels[x] = y;
        x = tmp;
    }
    
    return y;
}

// Union operation for union-find algorithm
__device__ void unionLabels(unsigned* labels, unsigned a, unsigned b) {
    unsigned rootA = find(labels, a);
    unsigned rootB = find(labels, b);
    
    if (rootA != rootB) {
        // Make the smaller label the representative
        if (rootA < rootB) {
            labels[rootB] = rootA;
        } else {
            labels[rootA] = rootB;
        }
    }
}

std::vector<unsigned char> loadPGM(const std::string& filename, int& width, int& height) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error: Unable to open file " << filename << std::endl;
        return {};
    }

    std::string magic;
    file >> magic;
    if (magic != "P5") {
        std::cerr << "Error: Invalid PGM file format." << std::endl;
        return {};
    }

    file >> width >> height;
    int max_val;
    file >> max_val;

    file.ignore(); // Ignore newline or carriage return before pixel data

    std::vector<unsigned char> pixels(width * height);
    file.read(reinterpret_cast<char*>(pixels.data()), pixels.size());

    file.close();

    return pixels;
}

//------------------------------------------------------------------------------
// Kernel 1: Strip Labeling
//------------------------------------------------------------------------------
__global__ void stripLabeling(
    const unsigned char* input,
    unsigned* labels,
    int width,
    int height,
    int pitch)
{
    // Each block processes a horizontal strip of 4 rows
    int blockStartRow = blockIdx.y * BLOCK_HEIGHT;
    int row = blockStartRow + threadIdx.y;
    int col = blockIdx.x * BLOCK_WIDTH + threadIdx.x;
    
    // Out of bounds check
    if (row >= height || col >= width) {
        return;
    }
    
    // Thread ID within the warp
    int tid = threadIdx.x;
    int warpId = threadIdx.y;
    
    // Shared memory for the bitmasks and temporary labels
    __shared__ unsigned bitmasks[BLOCK_HEIGHT];
    __shared__ unsigned temp_labels[BLOCK_HEIGHT][BLOCK_WIDTH];
    
    // Get the pixel value (0 or 1)
    unsigned char pixelValue = (col < width && row < height) ? input[row * pitch + col] > 0 : 0;
    
    // Build the bitmask for the current warp
    unsigned warpBitmask = buildBitmask(pixelValue, tid);
    
    // Store the bitmask in shared memory (only the first thread in each warp)
    if (tid == 0) {
        bitmasks[warpId] = warpBitmask;
    }
    
    __syncthreads();
    
    // Process only set pixels
    if (pixelValue) {
        unsigned startDist = start_distance(bitmasks[warpId], tid);
        
        // If this pixel is the start of a segment (startDist == 0)
        if (startDist == 0) {
            // Assign a new label based on global position
            unsigned label = row * width + col;
            temp_labels[warpId][tid] = label;
            
            // Initialize the label in the union-find data structure
            labels[label] = label;
        } else {
            // This pixel is part of an existing segment
            // Get the label from the start pixel of the segment
            temp_labels[warpId][tid] = temp_labels[warpId][tid - startDist];
        }
    } else {
        // For non-set pixels, assign invalid label
        temp_labels[warpId][tid] = UINT_MAX;
    }
    
    __syncthreads();
    
    // Write the labels to global memory for set pixels
    if (pixelValue) {
        labels[row * width + col] = temp_labels[warpId][tid];
    }
}

void previewImage(const std::vector<unsigned char>& image, int width, int height) {
    std::cout << "Image Preview:\n";
    
    int step_y = std::max(1, height / 64);
    int step_x = std::max(1, width / 64);

    for (int row = 0; row < height; row += step_y) {
        for (int col = 0; col < width; col += step_x) {
            char pixel = (image[row * width + col] > 0) ? '#' : ' ';
            std::cout << pixel;
        }
        std::cout << '\n';
    }
}

int main() {

    int width, height;
    std::vector<unsigned char> h_input = loadPGM("test_input.pgm", width, height);
    if (h_input.empty()) {
        return 1; // Error loading file
    }

    previewImage(h_input, width, height);

    // Allocate device memory and copy input data
    thrust::device_vector<unsigned char> d_input = h_input;
    thrust::device_vector<unsigned> d_labels(width * height, 0);

    // Launch CUDA kernel
    dim3 blocks((width + BLOCK_WIDTH - 1) / BLOCK_WIDTH, (height + BLOCK_HEIGHT - 1) / BLOCK_HEIGHT);
    dim3 threads(BLOCK_WIDTH, BLOCK_HEIGHT);

    std::cout<<"blocks size: "<<blocks.x<<", "<<blocks.y<<std::endl;
    std::cout<<"threads size: "<<threads.x<<", "<<threads.y<<std::endl;

    stripLabeling<<<blocks, threads>>>(thrust::raw_pointer_cast(d_input.data()),
                                       thrust::raw_pointer_cast(d_labels.data()),
                                       width, height, width);

    thrust::host_vector<unsigned> h_labels = d_labels;
    
    
    return 0;
}