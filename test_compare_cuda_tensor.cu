#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
using namespace nvcuda;
#include <iostream>
#include <random>
#include <utility>
#include <vector>
#include <hip/hip_fp16.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

void fill_random_float_values(float* arr, size_t n, std::default_random_engine& e)
{
    std::uniform_real_distribution<float> uniform_dist(-256, 256);
    for (size_t i{0}; i < n; ++i) {
        arr[i] = uniform_dist(e);
    }
}

void fill_random_int8_values(int8_t* arr, size_t n, std::default_random_engine& e)
{
    std::uniform_int_distribution<int8_t> uniform_dist(-128, 127);
    for (size_t i{0}; i < n; ++i) {
        arr[i] = uniform_dist(e);
    }
}

void fill_random_int16_values(int16_t* arr, size_t n, std::default_random_engine& e)
{
    std::uniform_int_distribution<int16_t> uniform_dist(-128, 127);
    for (size_t i{0}; i < n; ++i) {
        arr[i] = uniform_dist(e);
    }
}

void fill_random_int32_values(int32_t* arr, size_t n, std::default_random_engine& e)
{
    std::uniform_int_distribution<int32_t> uniform_dist(-128, 127);
    for (size_t i{0}; i < n; ++i) {
        arr[i] = uniform_dist(e);
    }
}

void fill_random_half_values(__half* arr, size_t n, std::default_random_engine& e)
{
    std::uniform_real_distribution<float> uniform_dist(0.0f, 127.0f);
    for (size_t i{0}; i < n; ++i) {
        arr[i] = __float2half(uniform_dist(e));
    }
}

void fill_random_double_values(double* arr, size_t n, std::default_random_engine& e)
{
    std::uniform_real_distribution<double> uniform_dist(-128, 127);
    for (size_t i{0}; i < n; ++i) {
        arr[i] = uniform_dist(e);
    }
}

// Naive GEMM
template<typename T>
__global__ void __launch_bounds__(1024) gemm_naive(const T *__restrict__ dA, const T *__restrict__ dB, float *__restrict__ dC, int M, int K, int N)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    float tmp = 0;

    if (row < M && col < N)
    {
        for (int s = 0; s < K; s++)
        {
            tmp += __half2float(dA[row * K + s]) * __half2float(dB[s * N + col]);
        }
        dC[row * N + col] = tmp;
    }
}

template<typename T>
__global__ void __launch_bounds__(1024) gemm_CUDA(float *__restrict__ c, const T *__restrict__ a, const T *__restrict__ b, int M, int N, int K) {
    
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int TILE_SIZE = 16;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int col = bx * TILE_SIZE + tx;
    const int row = by * TILE_SIZE + ty;

    __shared__ T SA[TILE_SIZE][TILE_SIZE];
    __shared__ T SB[TILE_SIZE][TILE_SIZE];

    float sum = 0;
    for (int k = 0; k < (K + TILE_SIZE - 1)/TILE_SIZE; ++k) {
        if (row < M && k * TILE_SIZE + tx < K) {
            SA[ty][tx] = a[row * K + k * TILE_SIZE + tx];
        } else {
            SA[ty][tx] = 0;
        }

        if (col < N && k * TILE_SIZE + ty < K) {
            SB[ty][tx] = b[col + (k * TILE_SIZE + ty) * N];
        } else {
            SB[ty][tx] = 0;
        }

        __syncthreads();

        for (int n_k = 0; n_k < TILE_SIZE; ++n_k) {
            sum += __half2float(SA[ty][n_k]) * __half2float(SB[n_k][tx]);
        }
        __syncthreads();
    }

    if (row < M && col < N) {
        c[row * N + col] = sum;
    }
}

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define WARP_SIZE 32

__host__ __device__ int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

__global__ void wmmaNaiveKernel(const half *__restrict__ A, const half *__restrict__ B, float *__restrict__ C, size_t M, size_t N, size_t K) {
    const size_t K_tiles = div_ceil(K, WMMA_K);

    const size_t warp_row = blockIdx.y * WMMA_M;
    const size_t warp_col = blockIdx.x * WMMA_N;

    if (warp_row >= M || warp_col >= N) {
        return;
    }

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> C_frag;

    wmma::fill_fragment(C_frag, 0.0f);

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> A_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> B_frag;

        wmma::load_matrix_sync(A_frag, A + warp_row * K + i * WMMA_K, K);
        wmma::load_matrix_sync(B_frag, B + warp_col + i * WMMA_K * N, N);

        wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);
    }

    wmma::store_matrix_sync(C + warp_row * N + warp_col, C_frag, N, wmma::mem_row_major);
}

int main() {

    int M = 2048;
    int K = 2048;
    int N = 2048;

    std::cout << "Matrix Sizes" << std::endl;
    std::cout << "M: " << M << std::endl;
    std::cout << "N: " << N << std::endl;
    std::cout << "K: " << K << std::endl;

    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine random_engine(seed);

    thrust::host_vector<__half> h_a_vec(M*K);
    thrust::host_vector<__half> h_b_vec(K*N);

    fill_random_half_values(h_a_vec.data(), h_a_vec.size(), random_engine);
    fill_random_half_values(h_b_vec.data(), h_b_vec.size(), random_engine);
    
    thrust::device_vector<__half> d_a_vec = h_a_vec;
    thrust::device_vector<__half> d_b_vec = h_b_vec;
    thrust::device_vector<float> d_c_vec(M*N);

    // CPU
    /*std::vector<int16_t> h_c_cpu(M*N, 0);

    auto cpu_start = std::chrono::high_resolution_clock::now();
    for (int r=0 ; r<M ; ++r) {
        for (int c=0; c<N ; ++c) {
            for (int k=0; k<K; ++k) {
                h_c_cpu[r * N + c] += h_a_vec[r * K + k] * h_b_vec[k * N + c];
            }
        }
    }
    auto cpu_end = std::chrono::high_resolution_clock::now();
    auto t_cpu = std::chrono::duration_cast<std::chrono::microseconds>(cpu_end - cpu_start).count();*/

    dim3 threadNum(16, 16);
    dim3 blockNum((M + threadNum.x - 1)/threadNum.x, (N + threadNum.y - 1)/threadNum.y);

    hipEvent_t cuda_start, cuda_end;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_end);

    const int numIterations = 1;
    float naive_totalTime = 0.0f;

    // 1. CUDA NAIVE
    for (int i = 0; i < numIterations; ++i) {
        hipEventRecord(cuda_start, 0);

        gemm_naive<__half><<<blockNum, threadNum>>>(d_a_vec.data().get(), d_b_vec.data().get(), d_c_vec.data().get(), M, K, N);

        hipEventRecord(cuda_end, 0);
        hipEventSynchronize(cuda_end);

        float ms = 0.0f;
        hipEventElapsedTime(&ms, cuda_start, cuda_end);

        naive_totalTime += ms;
    }

    thrust::host_vector<float> h_naive_c_vec = d_c_vec;

    // 2. CUDA
    float v2_totalTime = 0.0f;
    for (int i = 0; i < numIterations; ++i) {
        hipEventRecord(cuda_start, 0);

        gemm_CUDA<__half><<<blockNum, threadNum>>>(d_c_vec.data().get(), d_a_vec.data().get(), d_b_vec.data().get(), M, N, K);

        hipEventRecord(cuda_end, 0);
        hipEventSynchronize(cuda_end);

        float ms = 0.0f;
        hipEventElapsedTime(&ms, cuda_start, cuda_end);

        v2_totalTime += ms;
    }
    
    thrust::host_vector<float> h_c_vec = d_c_vec;

    // 3. 

    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(N, WMMA_N), div_ceil(M, WMMA_M));

    float v3_totalTime = 0.0f;
    for (int i = 0; i < numIterations; ++i) {
        hipEventRecord(cuda_start, 0);

        wmmaNaiveKernel<<<grid, block>>>(d_a_vec.data().get(), d_b_vec.data().get(), d_c_vec.data().get(), M, N, K);

        hipEventRecord(cuda_end, 0);
        hipEventSynchronize(cuda_end);

        float ms = 0.0f;
        hipEventElapsedTime(&ms, cuda_start, cuda_end);

        v3_totalTime += ms;
    }
    
    thrust::host_vector<float> h_c_vec_v3 = d_c_vec;

    // compare
    bool flg = 0;
    float cuda_error = 0.0f;
    float tensor_error = 0.0f;

    for (int r=0 ; r<M ; ++r) {
        for (int c=0; c<N ; ++c) {
            float naive_res = h_naive_c_vec[r * N + c];
            float cuda_res = h_c_vec[r * N + c];
            float tensor_res = h_c_vec_v3[r * N + c];

            float err_cuda = abs(naive_res-cuda_res)/naive_res;
            float err_tensor = abs(naive_res-tensor_res)/naive_res;

            cuda_error += err_cuda;
            tensor_error += err_tensor;

            if (err_cuda > 1e-3 || err_tensor > 1e-3) {
                printf("(%f, %f, %f)\n", h_naive_c_vec[r * N + c], h_c_vec[r * N + c], h_c_vec_v3[r * N + c]);
                printf("Failed: cuda, tensor: %f, %f\n", err_cuda, err_tensor);
                flg = 1;
            }
            if (flg) break;
        }
        if (flg) break;
    }

    if (!flg) {
        std::cout << "NAIVE execution time: " << naive_totalTime/numIterations << " ms" << std::endl;
        std::cout << "V2 execution time: " << v2_totalTime/numIterations << " ms, error: " << cuda_error/(M*N) << std::endl;
        std::cout << "V3 execution time: " << v3_totalTime/numIterations << " ms, error: " << tensor_error/(M*N) << std::endl;
    }

    return 0;
}