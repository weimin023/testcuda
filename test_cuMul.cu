#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <chrono>
#include "hip/hip_fp16.h"

#define N 500000

__device__ static __inline__ hipFloatComplex cuCmulfOpt(const hipFloatComplex a, const hipFloatComplex b) {
    return make_hipFloatComplex(__hsub(__hmul(a.x, b.x), __hmul(a.y, b.y)), __hfma(a.x, b.y, __hmul(a.y, b.x)));
    // __hfma(a.x, b.x, __hneg(__hmul(a.y, b.y)));
}

__global__ void kernel_cuCmulf(hipFloatComplex *out, const hipFloatComplex *a, const hipFloatComplex *b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = hipCmulf(a[idx], b[idx]);
    }
}

__global__ void kernel_cuCmulfOpt(hipFloatComplex *out, const hipFloatComplex *a, const hipFloatComplex *b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = cuCmulfOpt(a[idx], b[idx]);  // using out[idx] as a dummy third parameter
    }
}

void profileFunction(void (*kernel)(hipFloatComplex*, const hipFloatComplex*, const hipFloatComplex*), 
                     hipFloatComplex *d_out, const hipFloatComplex *d_a, const hipFloatComplex *d_b) {
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    // Launch the kernel multiple times to accumulate enough time for profiling
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 1; i++) {
        kernel<<<gridSize, blockSize>>>(d_out, d_a, d_b);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    std::cout << "Time taken: " << duration.count() / 100 << " ms per kernel call" << std::endl;
}

int main() {
    hipFloatComplex *h_a = new hipFloatComplex[N];
    hipFloatComplex *h_b = new hipFloatComplex[N];
    hipFloatComplex *h_out = new hipFloatComplex[N];

    // Initialize input arrays
    for (int i = 0; i < N; ++i) {
        h_a[i] = make_hipFloatComplex(1.0f, 2.0f);
        h_b[i] = make_hipFloatComplex(3.0f, 4.0f);
    }

    hipFloatComplex *d_a, *d_b, *d_out;
    hipMalloc(&d_a, N * sizeof(hipFloatComplex));
    hipMalloc(&d_b, N * sizeof(hipFloatComplex));
    hipMalloc(&d_out, N * sizeof(hipFloatComplex));

    hipMemcpy(d_a, h_a, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    std::cout << "Profiling cuCmulf:" << std::endl;
    profileFunction(kernel_cuCmulf, d_out, d_a, d_b);

    std::cout << "Profiling cuCmulfOpt:" << std::endl;
    profileFunction(kernel_cuCmulfOpt, d_out, d_a, d_b);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    delete[] h_a;
    delete[] h_b;
    delete[] h_out;

    return 0;
}