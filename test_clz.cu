#include <iostream>
#include <hip/hip_runtime.h>

__global__ void bitOperationsKernel(int *input, int *leadingZeros, int *firstSetBit, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int val = input[idx];
        leadingZeros[idx] = __clz(val);
        firstSetBit[idx] = __ffs(val);
    }
}

int main() {
    const int N = 5;
    int h_input[N] = {static_cast<int>(0x80000000), 0x00000001, 0x0000000F, 0x00010000, 0}; 
    int h_leadingZeros[N], h_firstSetBit[N];

    int *d_input, *d_leadingZeros, *d_firstSetBit;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_leadingZeros, N * sizeof(int));
    hipMalloc(&d_firstSetBit, N * sizeof(int));

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    bitOperationsKernel<<<1, N>>>(d_input, d_leadingZeros, d_firstSetBit, N);
    hipMemcpy(h_leadingZeros, d_leadingZeros, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_firstSetBit, d_firstSetBit, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_leadingZeros);
    hipFree(d_firstSetBit);

    std::cout << "Value\tLeading Zeros\tFirst Set Bit\n";
    for (int i = 0; i < N; i++) {
        std::cout << std::hex << "0x" << h_input[i] << "\t" 
                  << std::dec << h_leadingZeros[i] << "\t\t"
                  << h_firstSetBit[i] << "\n";
    }

    return 0;
}
