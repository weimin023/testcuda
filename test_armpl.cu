#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hip/hip_complex.h>
#include <chrono>
#include <thrust/host_vector.h>
#include "armpl.h"
#include <complex.h> 

#define N 8

#define CHECK_CUDA(call)                                                                                                                                                                               \
    do {                                                                                                                                                                                               \
        hipError_t err = call;                                                                                                                                                                        \
        if (err != hipSuccess) {                                                                                                                                                                      \
            fprintf(stderr, "Cuda failed with error code %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);                                                                    \
            exit(EXIT_FAILURE);                                                                                                                                                                        \
        }                                                                                                                                                                                              \
    } while (0)

#define CHECK_CUSOLVER(call)                                                                                                                                                                           \
    do {                                                                                                                                                                                               \
        hipsolverStatus_t stat = call;                                                                                                                                                                  \
        if (stat != HIPSOLVER_STATUS_SUCCESS) {                                                                                                                                                         \
            fprintf(stderr, "CuSolver failed with error code %s at line %d in file %s\n", cusolverGetErrorString(stat), __LINE__, __FILE__);                                                           \
            exit(EXIT_FAILURE);                                                                                                                                                                        \
        }                                                                                                                                                                                              \
    } while (0)

#define CHECK_CUBLAS(call)                                                                                                                                                                               \
    do {                                                                                                                                                                                               \
        hipblasStatus_t err = call;                                                                                                                                                                        \
        if (err != HIPBLAS_STATUS_SUCCESS) {                                                                                                                                                                      \
            fprintf(stderr, "cuBLAS failed with error code %s at line %d in file %s\n", cublasGetStatusName(err), __LINE__, __FILE__);                                                                    \
            exit(EXIT_FAILURE);                                                                                                                                                                        \
        }                                                                                                                                                                                              \
    } while (0)

const char* cusolverGetErrorString(hipsolverStatus_t status) {
    switch (status) {
        case HIPSOLVER_STATUS_SUCCESS:
            return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_MAPPING_ERROR:
            return "HIPSOLVER_STATUS_MAPPING_ERROR";
        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default:
            return "Unknown cuSOLVER error";
    }
}

hipFloatComplex h_cov[64] = {
        make_hipFloatComplex(4140.65f, -2.24197e-06f), make_hipFloatComplex(137.897f, 3938.01f),
        make_hipFloatComplex(-158.647f, -1241.86f), make_hipFloatComplex(-1936.52f, 2616.06f),
        make_hipFloatComplex(-743.871f, 2664.88f), make_hipFloatComplex(623.323f, -4050.98f),
        make_hipFloatComplex(-1115.17f, 1268.56f), make_hipFloatComplex(-1067.13f, 133.398f),
        make_hipFloatComplex(137.897f, -3938.01f), make_hipFloatComplex(4154.02f, 3.32665e-06f),
        make_hipFloatComplex(-1563.14f, -967.872f), make_hipFloatComplex(2961.04f, 2489.75f),
        make_hipFloatComplex(3325.31f, 1229.58f), make_hipFloatComplex(-3941.03f, -737.05f),
        make_hipFloatComplex(1463.98f, 2163.99f), make_hipFloatComplex(-995.137f, 1257.28f),
        make_hipFloatComplex(-158.647f, 1241.86f), make_hipFloatComplex(-1563.14f, 967.872f),
        make_hipFloatComplex(3843.31f, -2.39818e-07f), make_hipFloatComplex(-2763.36f, 255.503f),
        make_hipFloatComplex(-2805.64f, 1415.83f), make_hipFloatComplex(1326.26f, 228.303f),
        make_hipFloatComplex(-3621.11f, -722.38f), make_hipFloatComplex(584.441f, -3733.83f),
        make_hipFloatComplex(-1936.52f, -2616.06f), make_hipFloatComplex(2961.04f, -2489.75f),
        make_hipFloatComplex(-2763.36f, -255.503f), make_hipFloatComplex(4067.95f, -3.1887e-06f),
        make_hipFloatComplex(3742.5f, -1311.93f), make_hipFloatComplex(-2992.45f, 1599.3f),
        make_hipFloatComplex(3214.86f, 1165.23f), make_hipFloatComplex(-611.432f, 2481.71f),
        make_hipFloatComplex(-743.871f, -2664.88f), make_hipFloatComplex(3325.31f, -1229.58f),
        make_hipFloatComplex(-2805.64f, -1415.83f), make_hipFloatComplex(3742.5f, 1311.94f),
        make_hipFloatComplex(4021.9f, -3.66673e-06f), make_hipFloatComplex(-2973.91f, 342.678f),
        make_hipFloatComplex(2854.54f, 2360.54f), make_hipFloatComplex(-1725.97f, 2476.39f),
        make_hipFloatComplex(623.323f, 4050.98f), make_hipFloatComplex(-3941.03f, 737.05f),
        make_hipFloatComplex(1326.26f, -228.303f), make_hipFloatComplex(-2992.45f, -1599.3f),
        make_hipFloatComplex(-2973.91f, -342.678f), make_hipFloatComplex(4204.25f, 3.96076e-06f),
        make_hipFloatComplex(-1613.83f, -1059.15f), make_hipFloatComplex(-198.559f, -1266.46f),
        make_hipFloatComplex(-1115.17f, -1268.56f), make_hipFloatComplex(1463.98f, -2163.99f),
        make_hipFloatComplex(-3621.11f, 722.38f), make_hipFloatComplex(3214.86f, -1165.23f),
        make_hipFloatComplex(2854.54f, -2360.54f), make_hipFloatComplex(-1613.83f, 1059.15f),
        make_hipFloatComplex(3900.07f, -1.04732e-06f), make_hipFloatComplex(171.296f, 3648.69f),
        make_hipFloatComplex(-1067.13f, -133.398f), make_hipFloatComplex(-995.137f, -1257.28f),
        make_hipFloatComplex(584.441f, 3733.84f), make_hipFloatComplex(-611.432f, -2481.71f),
        make_hipFloatComplex(-1725.97f, -2476.39f), make_hipFloatComplex(-198.559f, 1266.46f),
        make_hipFloatComplex(171.296f, -3648.69f), make_hipFloatComplex(3863.89f, -2.28193e-06f)
    };

armpl_singlecomplex_t h_cov_arm[N * N] = {
    {4140.65f, -2.24197e-06f}, {137.897f, 3938.01f}, {-158.647f, -1241.86f}, {-1936.52f, 2616.06f},
    {-743.871f, 2664.88f}, {623.323f, -4050.98f}, {-1115.17f, 1268.56f}, {-1067.13f, 133.398f},
    {137.897f, -3938.01f}, {4154.02f, 3.32665e-06f}, {-1563.14f, -967.872f}, {2961.04f, 2489.75f},
    {3325.31f, 1229.58f}, {-3941.03f, -737.05f}, {1463.98f, 2163.99f}, {-995.137f, 1257.28f},
    {-158.647f, 1241.86f}, {-1563.14f, 967.872f}, {3843.31f, -2.39818e-07f}, {-2763.36f, 255.503f},
    {-2805.64f, 1415.83f}, {1326.26f, 228.303f}, {-3621.11f, -722.38f}, {584.441f, -3733.83f},
    {-1936.52f, -2616.06f}, {2961.04f, -2489.75f}, {-2763.36f, -255.503f}, {4067.95f, -3.1887e-06f},
    {3742.5f, -1311.93f}, {-2992.45f, 1599.3f}, {3214.86f, 1165.23f}, {-611.432f, 2481.71f},
    {-743.871f, -2664.88f}, {3325.31f, -1229.58f}, {-2805.64f, -1415.83f}, {3742.5f, 1311.94f},
    {4021.9f, -3.66673e-06f}, {-2973.91f, 342.678f}, {2854.54f, 2360.54f}, {-1725.97f, 2476.39f},
    {623.323f, 4050.98f}, {-3941.03f, 737.05f}, {1326.26f, -228.303f}, {-2992.45f, -1599.3f},
    {-2973.91f, -342.678f}, {4204.25f, 3.96076e-06f}, {-1613.83f, -1059.15f}, {-198.559f, -1266.46f},
    {-1115.17f, -1268.56f}, {1463.98f, -2163.99f}, {-3621.11f, 722.38f}, {3214.86f, -1165.23f},
    {2854.54f, -2360.54f}, {-1613.83f, 1059.15f}, {3900.07f, -1.04732e-06f}, {171.296f, 3648.69f},
    {-1067.13f, -133.398f}, {-995.137f, -1257.28f}, {584.441f, 3733.84f}, {-611.432f, -2481.71f},
    {-1725.97f, -2476.39f}, {-198.559f, 1266.46f}, {171.296f, -3648.69f}, {3863.89f, -2.28193e-06f}
};

void ver_armpl() {
    armpl_int_t n = N;
    armpl_int_t kd = N - 1;
    armpl_int_t ldab = N;

    float w[N];                      
    armpl_singlecomplex_t z[N * N];  


    armpl_int_t lwork = -1, lrwork = -1, liwork = -1, info;
    armpl_singlecomplex_t work_query;
    float rwork_query;

    chbevd_2stage_("V", "U", &n, &kd, h_cov_arm, &ldab, w, z, &n,
                   &work_query, &lwork, &rwork_query, &lrwork, NULL, &liwork, &info);

    if (info != 0) {
        // Handle error, such as logging or returning
        fprintf(stderr, "Error in workspace query: %d\n", info);
        return;
    }

    // Allocate workspace based on the query results
    lwork = (armpl_int_t)work_query; // Get the size of work
    lrwork = (armpl_int_t)rwork_query; // Get the size of rwork
    liwork = 5*N/* allocate size based on needs */; // Determine and allocate size for iwork

    // Allocate work, rwork, and iwork arrays
    armpl_singlecomplex_t *work = (armpl_singlecomplex_t *)malloc(lwork * sizeof(armpl_singlecomplex_t));
    float *rwork = (float *)malloc(lrwork * sizeof(float));
    armpl_int_t *iwork = (armpl_int_t *)malloc(liwork * sizeof(armpl_int_t));

    // Second call to actually perform the computation
    chbevd_2stage_("V", "U", &n, &kd, h_cov_arm, &ldab, w, z, &n,
                   work, &lwork, rwork, &lrwork, iwork, &liwork, &info);

    // Check for errors in the actual computation
    if (info != 0) {
        fprintf(stderr, "Error in computation: %d\n", info);
        free(work);
        free(rwork);
        free(iwork);
        return;
    }

    // If successful, print eigenvalues and eigenvectors
    printf("Eigenvalues:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", w[i]);
    }

    printf("Eigenvectors:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
	    //armpl_singlecomplex_t tmp = z[i * n + j];
            //printf("(%f, %f) ", tmp.real, tmp.imag);
        }
        //printf("\n");
    }

    // Free allocated memory
    free(work);
    free(rwork);
    free(iwork);
}

void ver_cuda(hipFloatComplex *m_eigen_vec, float *m_eigen_val, const hipFloatComplex *d_cov, hipsolverHandle_t &m_cusolverH) {
    hipFloatComplex *d_work;
    int *devInfo;
    int lwork = 0;

    CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnCheevd_bufferSize(m_cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N, m_eigen_vec, N, m_eigen_val, &lwork));
    CHECK_CUDA(hipMalloc(&d_work, lwork * sizeof(hipComplex)));

    CHECK_CUDA(hipMemcpy(m_eigen_vec, d_cov, N * N * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    
    auto start = std::chrono::high_resolution_clock::now();
    CHECK_CUSOLVER(hipsolverDnCheevd(m_cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, N, m_eigen_vec, N, m_eigen_val, d_work, lwork, devInfo));
    auto end = std::chrono::high_resolution_clock::now();

    int info;
    CHECK_CUDA(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (info != 0) {
        std::cerr << "Error: cuSOLVER operation failed with info = " << info << std::endl;
    }
}

int main() {
    hipsolverHandle_t m_cusolverH;
    hipsolverDnCreate(&m_cusolverH);

    hipFloatComplex *d_eigen_vec;
    float *d_eigen_val;
    
    hipMalloc((void**)&d_eigen_vec, N * N * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_eigen_val, N * sizeof(float));

    ver_cuda(d_eigen_vec, d_eigen_val, h_cov, m_cusolverH);

    thrust::host_vector<hipFloatComplex> h_eigen_vec(N*N);
    thrust::host_vector<float> h_eigen_val(N);
    hipMemcpy(h_eigen_vec.data(), d_eigen_vec, N * N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(h_eigen_val.data(), d_eigen_val, N * sizeof(float), hipMemcpyDeviceToHost);

    std:: cout << "=========CHECK EIGENVALS=========" << std::endl;
    for (int i=0;i<N;++i) {
        std::cout << h_eigen_val[i] << std::endl;
    }

    ver_armpl();

    return 0;
}
